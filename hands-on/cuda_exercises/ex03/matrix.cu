#include "hip/hip_runtime.h"
// C++ standard headers
#include <cassert>
#include <iostream>
#include <vector>

// CUDA headers
#include <hip/hip_runtime.h>

// local headers
#include "cuda_check.h"

// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Part 2 of 4: implement the kernel
__global__ void kernel(int* a, int dimx, int dimy)
{
  ___
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main()
{
  CUDA_CHECK(hipSetDevice(MYDEVICE));

  // Create a CUDA stream to execute asynchronous operations on this device
  hipStream_t queue;
  CUDA_CHECK(hipStreamCreate(&queue));

  // Part 1 and 4 of 4: set the dimensions of the matrix
  int dimx = ___;
  int dimy = ___;

  // Allocate enough memory on the host
  std::vector<int> h_a(dimx * dimy);
  int num_bytes = dimx * dimy * sizeof(int);

  // Pointer for the device memory
  int* d_a;

  // Allocate enough memory on the device
  CUDA_CHECK(hipMallocAsync(___));

  // Part 2 of 4: define grid and block size and launch the kernel
  dim3 grid, block;
  block.x = ___;
  block.y = ___;
  grid.x  = ___;
  grid.y  = ___;

  kernel<<<grid, block, 0, queue>>>(d_a, dimx, dimy);
  CUDA_CHECK(hipGetLastError());

  // Device to host copy
  CUDA_CHECK(hipMemcpyAsync(___));

  // Wait for all asynchronous operations to complete
  CUDA_CHECK(hipStreamSynchronize(queue));

  // Free the device memory
  CUDA_CHECK(hipFreeAsync(d_a, queue));

  // verify the data returned to the host is correct
  for (int row = 0; row < dimy; ++row) {
    for (int col = 0; col < dimx; ++col)
      assert(h_a[row * dimx + col] == row * dimx + col);
  }

  // Destroy the CUDA stream
  CUDA_CHECK(hipStreamDestroy(queue));

  // If the program makes it this far, then the results are correct and
  // there are no run-time errors.  Good work!
  std::cout << "Correct!" << std::endl;

  return 0;
}
